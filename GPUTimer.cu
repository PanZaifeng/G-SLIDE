#include "GPUTimer.h"


GPUTimer::GPUTimer() { 
    hipEventCreate(&start_time);
    hipEventCreate(&end_time);
}

GPUTimer::~GPUTimer() {
    hipEventDestroy(start_time);
    hipEventDestroy(end_time);
}

void GPUTimer::start() {
    hipEventRecord(start_time);
}

// return the elapsed tiem from start_time to end_time
float GPUTimer::record(std::string msg) {
    hipDeviceSynchronize();
    hipEventRecord(end_time);
    hipEventSynchronize(end_time);
    hipEventElapsedTime(&elapsed_time, start_time, end_time);

    if (msg != "")
        std::cout << msg << elapsed_time << " ms" << std::endl;
    
    // restart timer
    hipEventRecord(start_time);

    return elapsed_time;
}
