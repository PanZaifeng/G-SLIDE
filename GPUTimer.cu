#include "GPUTimer.h"
#include "utils.h"
#include <cstdio>

GPUTimer::GPUTimer() {
  CUDA_CHECK(hipEventCreate(&start_time));
  CUDA_CHECK(hipEventCreate(&end_time));
}

GPUTimer::~GPUTimer() {
  CUDA_CHECK(hipEventDestroy(start_time));
  CUDA_CHECK(hipEventDestroy(end_time));
}

void GPUTimer::start() {
  CUDA_CHECK(hipEventRecord(start_time));
}

// return the elapsed tiem from start_time to end_time
float GPUTimer::record(std::string msg) {
  CUDA_CHECK(hipDeviceSynchronize());
  CUDA_CHECK(hipEventRecord(end_time));
  CUDA_CHECK(hipEventSynchronize(end_time));
  CUDA_CHECK(hipEventElapsedTime(&elapsed_time, start_time, end_time));

  if (msg != "") {
    printf("%s%f ms\n", msg.c_str(), elapsed_time);
  }

  // restart timer
  CUDA_CHECK(hipEventRecord(start_time));

  return elapsed_time;
}
