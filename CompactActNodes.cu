#include "CompactActNodes.h"
#include "utils.h"


CompactActNodes::CompactActNodes(int batch_capacity, int node_capacity,
                                 bool is_managed)
: batch_capacity(batch_capacity), node_capacity(node_capacity) {
    if (!is_managed) {
        CUDA_CHECK( hipMalloc(&d_nodes, sizeof(int) * node_capacity) );
        CUDA_CHECK( hipMalloc(&d_vals, sizeof(float) * node_capacity) );
        CUDA_CHECK( hipMalloc(&d_cols, sizeof(int) * (batch_capacity + 1)) );
    } else {
        CUDA_CHECK( hipMallocManaged(&d_nodes,
                        sizeof(int) * node_capacity) );
        CUDA_CHECK( hipMallocManaged(&d_vals,
                        sizeof(float) * node_capacity) );
        CUDA_CHECK( hipMallocManaged(&d_cols,
                        sizeof(int) * (batch_capacity + 1)) );
    }
    
    CUDA_CHECK( hipMemset(d_nodes, 0, sizeof(int) * node_capacity) );
    CUDA_CHECK( hipMemset(d_vals, 0, sizeof(float) * node_capacity) );
    CUDA_CHECK( hipMemset(d_cols, 0, sizeof(int) * (batch_capacity + 1)) );
}

CompactActNodes::~CompactActNodes() {
    CUDA_CHECK( hipFree(d_nodes) );
    CUDA_CHECK( hipFree(d_vals) );
    CUDA_CHECK( hipFree(d_cols) );
}

void CompactActNodes::extract_from(const std::vector<int> &h_c_nodes,
                                   const std::vector<float> &h_c_vals,
                                   const std::vector<int> &h_c_cols)
{
    // printf("[%d, %d]\n", (int) h_c_nodes.size(), node_capacity);
    // assert(h_c_nodes.size() <= node_capacity);

    CUDA_CHECK( hipMemcpy(d_nodes, &h_c_nodes[0],
                    sizeof(int) * h_c_nodes.size(), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(d_vals, &h_c_vals[0],
                    sizeof(float) * h_c_vals.size(), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(d_cols, &h_c_cols[0],
                    sizeof(int) * h_c_cols.size(), hipMemcpyHostToDevice) );
}

void CompactActNodes::extract_from(const std::vector<int> &h_c_nodes,
                                   const std::vector<int> &h_c_cols)
{
    // printf("[%d, %d]\n", (int) h_c_nodes.size(), node_capacity);
    // assert(h_c_nodes.size() <= node_capacity);

    CUDA_CHECK( hipMemcpy(d_nodes, &h_c_nodes[0],
                    sizeof(int) * h_c_nodes.size(), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(d_cols, &h_c_cols[0],
                    sizeof(int) * h_c_cols.size(), hipMemcpyHostToDevice) );
}

void CompactActNodes::extract_to(std::vector<int> &h_c_nodes,
                                 std::vector<float> &h_c_vals,
                                 std::vector<int> &h_c_cols,
                                 const int batch_size)
{
    h_c_cols = std::vector<int>(batch_size + 1);
    CUDA_CHECK( hipMemcpy(&h_c_cols[0], d_cols,
                    sizeof(int) * h_c_cols.size(), hipMemcpyDeviceToHost) );
    
    int c_size = h_c_cols.back();
    h_c_nodes = std::vector<int>(c_size);
    h_c_vals = std::vector<float>(c_size);
    CUDA_CHECK( hipMemcpy(&h_c_nodes[0], d_nodes, sizeof(int) * c_size,
                    hipMemcpyDeviceToHost) );
    CUDA_CHECK( hipMemcpy(&h_c_vals[0], d_vals, sizeof(float) * c_size,
                    hipMemcpyDeviceToHost) );
}
