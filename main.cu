#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <string>
#include <vector>

#include "GPUTimer.h"
#include "Network.h"
#include "jsoncpp/json/json.h"
#include "utils.h"

std::vector<int> jarr_to_vec(const Json::Value &jarr) {
  const size_t size = jarr.size();
  std::vector<int> res(size);
  for (int i = 0; i < size; ++i) {
    res[i] = jarr[i].asInt();
  }

  return res;
}

// Return: real batch size
int get_batch_data(std::ifstream &ist, std::vector<int> &h_cmprs_input_nodes,
                   std::vector<float> &h_cmprs_input_vals,
                   std::vector<int> &h_cmprs_input_offsets,
                   std::vector<int> &h_cmprs_labels,
                   std::vector<int> &h_cmprs_label_offsets,
                   const int batch_size) {
  h_cmprs_input_nodes.clear();
  h_cmprs_input_vals.clear();
  h_cmprs_input_offsets.clear();
  h_cmprs_labels.clear();
  h_cmprs_label_offsets.clear();

  h_cmprs_input_offsets.push_back(0);
  h_cmprs_label_offsets.push_back(0);

  for (int b = 0; b < batch_size; ++b) {
    int label;
    if (ist >> label) {
      h_cmprs_labels.push_back(label);
    } else {
      return b;
    }

    while (ist.get() == ',') {
      ist >> label;
      h_cmprs_labels.push_back(label);
    }
    h_cmprs_label_offsets.push_back(h_cmprs_labels.size());

    do {
      int node;
      ist >> node;
      assert(ist.get() == ':');

      float val;
      ist >> val;
      h_cmprs_input_nodes.push_back(node);
      h_cmprs_input_vals.push_back(val);
    } while (ist.get() == ' ');
    h_cmprs_input_offsets.push_back(h_cmprs_input_nodes.size());
  }

  return batch_size;
}

int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("Usage: %s config.json\n", argv[0]);
    exit(1);
  }

  Json::Reader reader;
  Json::Value root;
  std::ifstream config_ist(argv[1]);
  if (!reader.parse(config_ist, root)) {
    printf("Parse %s failed!\n", argv[1]);
    exit(1);
  }

  const std::vector<int> node_num_per_layer =
      jarr_to_vec(root["node_num_per_layer"]);
  const std::vector<int> node_capacity_per_layer =
      jarr_to_vec(root["node_capacity_per_layer"]);
  const int input_size = root["input_size"].asInt();
  const int max_batch_size = root["max_batch_size"].asInt();
  const int input_capacity = root["input_capacity"].asInt();
  const int label_capacity = root["label_capacity"].asInt();
  const int K = root["K"].asInt(), L = root["L"].asInt();
  const int bin_size = root["bin_size"].asInt();
  const int bucket_num_per_tbl = root["bucket_num_per_tbl"].asInt();
  const int bucket_capacity = root["bucket_capacity"].asInt();
  const int threshold = root["threshold"].asInt();
  const int tbl_num_per_tile = root["tbl_num_per_tile"].asInt();
  const int tbl_num_per_thread = root["tbl_num_per_thread"].asInt();
  const int linked_bucket_num_per_tbl =
      root["linked_bucket_num_per_tbl"].asInt();
  const int linked_pool_size = root["linked_pool_size"].asInt();

  Network network(node_num_per_layer, node_capacity_per_layer, input_size,
                  max_batch_size, input_capacity, label_capacity, K, L,
                  bin_size, bucket_num_per_tbl, bucket_capacity, threshold,
                  tbl_num_per_tile, tbl_num_per_thread,
                  linked_bucket_num_per_tbl, linked_pool_size);

  const std::vector<int> max_act_nums = jarr_to_vec(root["max_act_nums"]);
  const int max_label_num = root["max_label_num"].asInt();
  const float lr = root["lr"].asFloat();
  const float BETA1 = root["BETA1"].asFloat();
  const float BETA2 = root["BETA2"].asFloat();
  const int rebuild_period = root["rebuild_period"].asInt();
  const int thread_num = root["thread_num"].asInt();
  const int epoch_num = root["epoch_num"].asInt();

  const std::string train_fname = root["train_fname"].asString();
  const std::string test_fname = root["test_fname"].asString();

  GPUTimer timer;
  float tot_time = 0;

  for (int e = 0; e < epoch_num; e++) {
    printf("------------------- Epoch %d ---------------------\n", e);
    std::ifstream train_ist(train_fname);
    std::ifstream test_ist(test_fname);
    if (!train_ist || !test_ist) {
      std::cerr << "Cannot open dataset file!" << std::endl;
      exit(-1);
    }

    std::string header;
    std::getline(train_ist, header);  // skip header
    std::getline(test_ist, header);   // skip header

    int batch_size;
    int cnt = 0;
    do {
      std::vector<int> h_cmprs_input_nodes;
      std::vector<float> h_cmprs_input_vals;
      std::vector<int> h_cmprs_input_offsets;
      std::vector<int> h_cmprs_labels;
      std::vector<int> h_cmprs_label_offsets;
      batch_size =
          get_batch_data(train_ist, h_cmprs_input_nodes, h_cmprs_input_vals,
                         h_cmprs_input_offsets, h_cmprs_labels,
                         h_cmprs_label_offsets, max_batch_size);

      const float tmplr =
          lr * sqrt((1 - pow(BETA2, cnt + 1))) / (1 - pow(BETA1, cnt + 1));
      const bool rebuild = cnt % 5 == 4;

      timer.start();

      network.train(h_cmprs_input_nodes, h_cmprs_input_vals,
                    h_cmprs_input_offsets, h_cmprs_labels,
                    h_cmprs_label_offsets, max_act_nums, batch_size, tmplr,
                    max_label_num, thread_num, rebuild);

      tot_time += timer.record("[BATCH " + std::to_string(cnt) + "] ");

      cnt++;
      // if (cnt > 10) break;

    } while (batch_size == max_batch_size);
    network.rebuild();

    printf("Current elapsed time %f ms\n", tot_time);

    // eval
    int correct_cnt = 0, test_cnt = 0;
    do {
      std::vector<int> h_cmprs_input_nodes;
      std::vector<float> h_cmprs_input_vals;
      std::vector<int> h_cmprs_input_offsets;
      std::vector<int> h_cmprs_labels;
      std::vector<int> h_cmprs_label_offsets;
      batch_size =
          get_batch_data(test_ist, h_cmprs_input_nodes, h_cmprs_input_vals,
                         h_cmprs_input_offsets, h_cmprs_labels,
                         h_cmprs_label_offsets, max_batch_size);

      timer.start();

      correct_cnt += network.eval(
          h_cmprs_input_nodes, h_cmprs_input_vals, h_cmprs_input_offsets,
          h_cmprs_labels, h_cmprs_label_offsets, batch_size, thread_num);

      // timer.record("Infer time ");

      test_cnt += batch_size;
      // if (test_cnt >= 512) break;

    } while (batch_size == max_batch_size);

    printf("Test %d records, %d correct; accuracy: %f\n", test_cnt, correct_cnt,
           ((float)correct_cnt) / test_cnt);
  }
}
