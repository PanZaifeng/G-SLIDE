#include <thrust/execution_policy.h>
#include <thrust/sort.h>

#include <unordered_map>
#include <vector>

#include "LSH.h"
#include "lshKnl.h"
#include "utils.h"

LSH::LSH(const int node_num, const int prev_node_num, const int max_batch_size,
         const int K, const int L, const int bin_size,
         const int bucket_num_per_tbl, const int bucket_capacity,
         const int threshold, const int tbl_num_per_tile,
         const int tbl_num_per_thread, const int linked_bucket_num_per_tbl,
         const int linked_pool_size)
    : node_num(node_num),
      prev_node_num(prev_node_num),
      K(K),
      L(L),
      bin_size(bin_size),
      bucket_num_per_tbl(bucket_num_per_tbl),
      bucket_capacity(bucket_capacity),
      tot_elem_num(K * L * bin_size),
      tbl_num_per_tile(tbl_num_per_tile),
      tbl_num_per_thread(tbl_num_per_thread),
      cmprs_gathered(L * max_batch_size, bucket_capacity * L * max_batch_size,
                     false, true),
      multi_linked_htables(max_batch_size, linked_bucket_num_per_tbl,
                           linked_pool_size, threshold) {
  CUDA_CHECK(
      hipMallocManaged(&d_rand_keys, sizeof(unsigned int) * tot_elem_num));
  CUDA_CHECK(hipMalloc(&d_bins, sizeof(int) * tot_elem_num));

  const int thread_num = 128;
  const int block_num = (tot_elem_num + thread_num - 1) / thread_num;
  init_bins_knl<<<block_num, thread_num>>>(d_bins, prev_node_num, tot_elem_num);

  const size_t tot_bucket_num = L * bucket_num_per_tbl;
  const size_t tot_bucket_capacity = tot_bucket_num * bucket_capacity;
  CUDA_CHECK(hipMallocManaged(&d_buckets, sizeof(int) * tot_bucket_capacity));
  CUDA_CHECK(hipMallocManaged(&d_bucket_sizes, sizeof(int) * tot_bucket_num));

  CUDA_CHECK(hipMallocManaged(&d_hashed_bucket_ids_colmajor,
                               sizeof(int) * L * max_batch_size));

  // CUDA_CHECK(hipMallocManaged(
  //     &d_gathered_nodes, sizeof(int) * bucket_capacity * L *
  //     max_batch_size));
  // CUDA_CHECK(hipMallocManaged(&d_gathered_offsets,
  //                              sizeof(int) * (1 + L * max_batch_size)));
  // CUDA_CHECK(hipMemset(d_gathered_offsets, 0, sizeof(int)));
}

LSH::~LSH() {
  CUDA_CHECK(hipFree(d_rand_keys));
  CUDA_CHECK(hipFree(d_bins));

  CUDA_CHECK(hipFree(d_buckets));
  CUDA_CHECK(hipFree(d_bucket_sizes));

  CUDA_CHECK(hipFree(d_hashed_bucket_ids_colmajor));

  // CUDA_CHECK(hipFree(d_gathered_nodes));
  // CUDA_CHECK(hipFree(d_gathered_offsets));
  cmprs_gathered.free();
  multi_linked_htables.free();
}

void LSH::shuffle_bins() {
  const int thread_num = 128;
  const int block_num = (tot_elem_num + thread_num - 1) / thread_num;
  gen_rand_keys_knl<<<block_num, thread_num>>>(d_rand_keys, rand(),
                                               prev_node_num, tot_elem_num);

  thrust::sort_by_key(thrust::device, d_rand_keys, d_rand_keys + tot_elem_num,
                      d_bins);
}

void LSH::build(const float *d_weights_rowmajor) {
  shuffle_bins();
  CUDA_CHECK(
      hipMemset(d_bucket_sizes, 0, sizeof(int) * L * bucket_num_per_tbl));

  const int thread_num = 64;
  const int block_num = (node_num + thread_num - 1) / thread_num;
  const int smem_size =
      (K * bin_size * tbl_num_per_tile + thread_num * prev_node_num) *
      sizeof(int);
  if (tbl_num_per_thread == tbl_num_per_tile) {
    init_hash_knl<<<block_num, thread_num, smem_size>>>(
        d_bins, d_weights_rowmajor, prev_node_num, node_num, tot_elem_num, L, K,
        bin_size, tbl_num_per_tile, bucket_num_per_tbl, bucket_capacity,
        d_buckets, d_bucket_sizes);
  } else {
    init_hash_knl<<<block_num, thread_num, smem_size>>>(
        d_bins, d_weights_rowmajor, prev_node_num, node_num, tot_elem_num, L, K,
        bin_size, tbl_num_per_tile, tbl_num_per_thread, bucket_num_per_tbl,
        bucket_capacity, d_buckets, d_bucket_sizes);
  }
}

void LSH::query_act_nodes(const CscActNodes &csc_inputs,
                          const CscActNodes &cmprs_labels, const int batch_size,
                          CscActNodes &csc_acts) {
  // Assume inputs is dense
  // TODO: dense -> sparse transform
  const float *d_dense_inputs_colmajor = csc_inputs.d_vals;

  const int thread_num = 128;
  const int hash_block_num = (L + tbl_num_per_tile - 1) / tbl_num_per_tile;
  const int smem_size = sizeof(int) * K * bin_size * tbl_num_per_tile;
  get_hash_knl<<<hash_block_num, thread_num, smem_size>>>(
      d_bins, d_dense_inputs_colmajor, d_bucket_sizes, prev_node_num,
      tot_elem_num, L, K, bin_size, tbl_num_per_tile, batch_size,
      bucket_num_per_tbl, bucket_capacity, d_hashed_bucket_ids_colmajor,
      cmprs_gathered.d_offsets + 1);

  thrust::inclusive_scan(thrust::device, cmprs_gathered.d_offsets + 1,
                         cmprs_gathered.d_offsets + 1 + L * batch_size,
                         cmprs_gathered.d_offsets + 1);

  const int gather_block_num = (L * batch_size + thread_num - 1) / thread_num;
  gather_buckets_knl<<<gather_block_num, thread_num>>>(
      d_hashed_bucket_ids_colmajor, d_buckets, L, batch_size,
      bucket_num_per_tbl, bucket_capacity, cmprs_gathered);

  multi_linked_htables.init_tbls();
  multi_linked_htables.block_reduce_cnt(cmprs_gathered, L, batch_size,
                                        thread_num);
  multi_linked_htables.activate_labels_seq(cmprs_labels, batch_size,
                                           thread_num);
  multi_linked_htables.get_act_nodes(csc_acts, batch_size);

  /*
  std::vector<int> h_gathered_nodes;
  std::vector<int> h_gathered_offsets;
  cmprs_gathered.extract_to(h_gathered_nodes, h_gathered_offsets, L *
  batch_size);

  std::vector<std::unordered_map<int, int>> golden_maps(batch_size);
  for (int i = 0; i < batch_size; ++i) {
    int begin = h_gathered_offsets[i * L];
    int end = h_gathered_offsets[(i + 1) * L];
    for (int j = begin; j < end; ++j) {
      int node = h_gathered_nodes[j];
      ++golden_maps[i][node];
    }
  }

  std::vector<int> h_labels;
  std::vector<int> h_label_offsets;
  cmprs_labels.extract_to(h_labels, h_label_offsets, batch_size);

  for (int i = 0; i < batch_size; ++i) {
    int begin = h_label_offsets[i];
    int end = h_label_offsets[i + 1];
    for (int j = begin; j < end; ++j) {
      int node = h_labels[j];
      ++golden_maps[i][node];
    }
  }

  std::vector<int> h_cmprs_nodes;
  std::vector<int> h_cmprs_offsets;
  csc_acts.extract_to(h_cmprs_nodes, h_cmprs_offsets, batch_size);

  bool pass = true;
  for (int i = 0; i < batch_size; ++i) {
    int begin = h_cmprs_offsets[i];
    int end = h_cmprs_offsets[i + 1];
    if (end - begin != golden_maps[i].size()) {
      printf("Size err at %d, device %d, golden %ld\n", i, end - begin,
             golden_maps[i].size());
      pass = false;
    } else {
      for (int j = begin; j < end; ++j) {
        int node = h_cmprs_nodes[j];
        if (!golden_maps[i].count(node)) {
          printf("Node err %d at %d\n", node, i);
          pass = false;
        }
      }
    }
  }

  if (pass) {
    printf("Query Pass!\n");
  } else {
    printf("Query Fail!\n");
    exit(-1);
  }
  */
}

void LSH::query_act_nodes(const CscActNodes &csc_inputs, const int batch_size,
                          CscActNodes &csc_acts) {
  // Assume inputs is dense
  // TODO: dense -> sparse transform
  const float *d_dense_inputs_colmajor = csc_inputs.d_vals;

  const int thread_num = 128;
  const int hash_block_num = (L + tbl_num_per_tile - 1) / tbl_num_per_tile;
  const int smem_size = sizeof(int) * K * bin_size * tbl_num_per_tile;
  get_hash_knl<<<hash_block_num, thread_num, smem_size>>>(
      d_bins, d_dense_inputs_colmajor, d_bucket_sizes, prev_node_num,
      tot_elem_num, L, K, bin_size, tbl_num_per_tile, batch_size,
      bucket_num_per_tbl, bucket_capacity, d_hashed_bucket_ids_colmajor,
      cmprs_gathered.d_offsets + 1);

  thrust::inclusive_scan(thrust::device, cmprs_gathered.d_offsets + 1,
                         cmprs_gathered.d_offsets + 1 + L * batch_size,
                         cmprs_gathered.d_offsets + 1);

  const int gather_block_num = (L * batch_size + thread_num - 1) / thread_num;
  gather_buckets_knl<<<gather_block_num, thread_num>>>(
      d_hashed_bucket_ids_colmajor, d_buckets, L, batch_size,
      bucket_num_per_tbl, bucket_capacity, cmprs_gathered);

  multi_linked_htables.init_tbls();
  multi_linked_htables.block_reduce_cnt(cmprs_gathered, L, batch_size,
                                        thread_num);
  multi_linked_htables.get_act_nodes(csc_acts, batch_size);
}