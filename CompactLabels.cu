#include "CompactLabels.h"
#include "utils.h"


CompactLabels::CompactLabels(int batch_capacity, int label_capacity)
: batch_capacity(batch_capacity), label_capacity(label_capacity) {
    CUDA_CHECK( hipMalloc(&d_nodes, sizeof(int) * label_capacity) );
    CUDA_CHECK( hipMalloc(&d_cols, sizeof(int) * (batch_capacity + 1)) );

    CUDA_CHECK( hipMemset(d_nodes, 0, sizeof(int) * label_capacity) );
    CUDA_CHECK( hipMemset(d_cols, 0, sizeof(int) * (batch_capacity + 1)) );
}

CompactLabels::~CompactLabels() {
    CUDA_CHECK( hipFree(d_nodes) );
    CUDA_CHECK( hipFree(d_cols) );
}

void CompactLabels::extract_from(const std::vector<int> &h_c_nodes,
                                 const std::vector<int> &h_c_cols)
{
    // printf("[%d, %d]\n", (int) h_c_nodes.size(), label_capacity);

    CUDA_CHECK( hipMemcpy(d_nodes, &h_c_nodes[0],
                    sizeof(int) * h_c_nodes.size(), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(d_cols, &h_c_cols[0],
                    sizeof(int) * h_c_cols.size(), hipMemcpyHostToDevice) );
}
